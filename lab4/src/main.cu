#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <filesystem>
#include <sstream>
#include <windows.h>

namespace fs = std::filesystem;

using namespace std;
using namespace std::chrono;

// Чтение матрицы из файла
vector<vector<int>> readMatrix(const string& filename) {
    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Ошибка открытия файла: " << filename << endl;
        exit(1);
    }

    vector<vector<int>> matrix;
    string line;

    while (getline(file, line)) {
        vector<int> row;
        stringstream ss(line);
        int value;

        while (ss >> value) {
            row.push_back(value);
        }

        matrix.push_back(row);
    }

    file.close();
    return matrix;
}

// Запись матрицы в файл
void writeMatrix(const string& filename, const vector<vector<int>>& matrix) {
    ofstream file(filename);
    if (!file.is_open()) {
        cerr << "Ошибка открытия файла: " << filename << endl;
        return;
    }

    for (const auto& row : matrix) {
        for (const auto& elem : row) {
            file << elem << " ";
        }
        file << endl;
    }

    file.close();
}

// CUDA kernel для умножения матриц
__global__ void matrixMulKernel(int* A, int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Обёртка для вызова CUDA-ядра
vector<vector<int>> multiplyMatricesCUDA(const vector<vector<int>>& A, const vector<vector<int>>& B) {
    int N = A.size();

    size_t mem_size = N * N * sizeof(int);

    // Хост-память
    int* h_A = new int[N * N];
    int* h_B = new int[N * N];
    int* h_C = new int[N * N];

    // Копирование матриц в одномерный массив
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j) {
            h_A[i * N + j] = A[i][j];
            h_B[i * N + j] = B[i][j];
        }

    // Устройство — GPU
    int* d_A, * d_B, * d_C;
    hipMalloc(&d_A, mem_size);
    hipMalloc(&d_B, mem_size);
    hipMalloc(&d_C, mem_size);

    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size, hipMemcpyHostToDevice);

    // Настройка сетки и блоков
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Запуск ядра
    matrixMulKernel << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C, N);
    hipDeviceSynchronize();  // Ждём завершения ядра

    hipMemcpy(h_C, d_C, mem_size, hipMemcpyDeviceToHost);

    // Освобождение ресурсов
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;

    // Копируем в векторный результат
    vector<vector<int>> result(N, vector<int>(N));
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            result[i][j] = h_C[i * N + j];

    delete[] h_C;

    return result;
}

int main() {
    SetConsoleOutputCP(65001);

    int sizes[] = { 50, 100, 150, 250, 500, 1000 };

    fs::path resultsDir = fs::current_path().parent_path() / "results";
    if (!fs::exists(resultsDir)) fs::create_directories(resultsDir);

    for (int size : sizes) {
        string fileA = "../data/matrix_A_" + to_string(size) + ".txt";
        string fileB = "../data/matrix_B_" + to_string(size) + ".txt";
        string resultFile = resultsDir.string() + "/result_" + to_string(size) + ".txt";

        auto matrixA = readMatrix(fileA);
        auto matrixB = readMatrix(fileB);

        auto start = high_resolution_clock::now();
        auto resultMatrix = multiplyMatricesCUDA(matrixA, matrixB);
        auto end = high_resolution_clock::now();

        writeMatrix(resultFile, resultMatrix);

        auto duration = duration_cast<milliseconds>(end - start).count();
        cout << "Matrix size: " << size << "x" << size << endl;
        cout << "Execution time: " << duration << " ms" << endl;
        cout << "Task size: " << size * size << " elements" << endl;
        cout << endl;
    }

    return 0;
}